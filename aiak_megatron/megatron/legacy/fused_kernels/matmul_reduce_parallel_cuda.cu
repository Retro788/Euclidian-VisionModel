#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <torch/torch.h>
#include <torch/csrc/distributed/c10d/ProcessGroupNCCL.hpp>

#define DIVUP(x, y) (((x) + (y)-1) / (y))

#define CUBLASCHECK(cmd)                                 \
    do {                                                 \
        hipblasStatus_t e = cmd;                          \
        if (e != HIPBLAS_STATUS_SUCCESS) {                \
            printf("Failed: CUBLAS error %s: %d '%d'\n", \
                   __FILE__,                             \
                   __LINE__,                             \
                   cmd);                                 \
            assert(false);                               \
        }                                                \
    } while (0)

#define CUDACHECK(cmd)                                \
    do {                                              \
        hipError_t e = cmd;                          \
        if (e != hipSuccess) {                       \
            printf("Failed: Cuda error %s:%d '%s'\n", \
                   __FILE__,                          \
                   __LINE__,                          \
                   hipGetErrorString(e));            \
            exit(EXIT_FAILURE);                       \
        }                                             \
    } while (0)

#define NCCLCHECK(cmd)                                \
    do {                                              \
        ncclResult_t r = cmd;                         \
        if (r != ncclSuccess) {                       \
            printf("Failed, NCCL error %s:%d '%s'\n", \
                   __FILE__,                          \
                   __LINE__,                          \
                   ncclGetErrorString(r));            \
            exit(EXIT_FAILURE);                       \
        }                                             \
    } while (0)

void cublasProcessMatMulChunk(hipblasHandle_t handle,
                              hipStream_t stream,
                              size_t chunkStart,
                              size_t chunkEnd,
                              const half* alpha,
                              const half* beta,
                              const at::Half* input,
                              const at::Half* weight,
                              at::Half* output,
                              int M,
                              int N,
                              int K,
                              bool column_parallel) {
    int rows = DIVUP(chunkEnd - chunkStart, N);
    int startRow = DIVUP(chunkStart, N);
    if (chunkStart == 0) {
        rows = min(rows, M);
    } else {
        if (M < startRow + rows) {
            rows = M - startRow;
        }
    }

    if (rows <= 0 || startRow >= M || rows > M || startRow + rows > M) {
        return;
    }

    if (column_parallel) {
        CUBLASCHECK(hipblasGemmEx(handle,
                                 HIPBLAS_OP_N,
                                 HIPBLAS_OP_N,
                                 N,
                                 rows,
                                 K,
                                 alpha,
                                 weight,
                                 HIP_R_16F,
                                 N,
                                 input + startRow * K,
                                 HIP_R_16F,
                                 K,
                                 beta,
                                 output + startRow * N,
                                 HIP_R_16F,
                                 N,
                                 HIP_R_16F,
                                 CUBLAS_GEMM_DFALT_TENSOR_OP));
    } else {
        CUBLASCHECK(hipblasGemmEx(handle,
                                 HIPBLAS_OP_T,
                                 HIPBLAS_OP_N,
                                 N,
                                 rows,
                                 K,
                                 alpha,
                                 weight,
                                 HIP_R_16F,
                                 K,
                                 input + startRow * K,
                                 HIP_R_16F,
                                 K,
                                 beta,
                                 output + startRow * N,
                                 HIP_R_16F,
                                 N,
                                 HIP_R_16F,
                                 CUBLAS_GEMM_DFALT_TENSOR_OP));
    }
}

void cublasProcessMatMulChunk(hipblasHandle_t handle,
                                hipStream_t stream,
                                size_t chunkStart,
                                size_t chunkEnd,
                                const float* alpha,
                                const float* beta,
                                const float* input,
                                const float* weight,
                                float* output,
                                int M,
                                int N,
                                int K,
                                bool column_parallel) {
    int rows = DIVUP(chunkEnd - chunkStart, N);
    int startRow = DIVUP(chunkStart, N);
    if (chunkStart == 0) {
        rows = min(rows, M);
    } else {
        if (M < startRow + rows) {
            rows = M - startRow;
        }
    }

    if (rows <= 0 || startRow >= M || rows > M || startRow + rows > M) {
        return;
    }

    if (column_parallel) {
        CUBLASCHECK(hipblasGemmEx(handle,
                        HIPBLAS_OP_N,
                        HIPBLAS_OP_N,
                        N,
                        rows,
                        K,
                        alpha,
                        weight,
                        HIP_R_32F,
                        N,
                        input + startRow * K,
                        HIP_R_32F,
                        K,
                        beta,
                        output + startRow * N,
                        HIP_R_32F,
                        N,
                        HIP_R_32F,
                        CUBLAS_GEMM_DFALT_TENSOR_OP));
    } else {
        CUBLASCHECK(hipblasGemmEx(handle,
                        HIPBLAS_OP_T,
                        HIPBLAS_OP_N,
                        N,
                        rows,
                        K,
                        alpha,
                        weight,
                        HIP_R_32F,
                        K,
                        input + startRow * K,
                        HIP_R_32F,
                        K,
                        beta,
                        output + startRow * N,
                        HIP_R_32F,
                        N,
                        HIP_R_32F,
                        CUBLAS_GEMM_DFALT_TENSOR_OP));
    }
}


int pipe_overlapped_with_split_kernel(hipblasHandle_t handle,
                                      const at::Half* input,
                                      const at::Half* weight,
                                      at::Half* output,
                                      int slit_n,
                                      int M,
                                      int N,
                                      int K,
                                      hipStream_t cublasStream,
                                      ncclComm_t comm,
                                      hipStream_t ncclStream,
                                      float alpha,
                                      float beta,
                                      bool column_parallel) {
    const half hAlpha = __float2half_rn(alpha);
    const half hBeta = __float2half_rn(beta);

    size_t M_N = static_cast<size_t>(M) * static_cast<size_t>(N);

    if (slit_n >= 0 && M_N % slit_n == 0) {
        
        size_t chunkSize = M_N / slit_n;
        hipEvent_t gemm_event;
        hipEventCreate(&gemm_event);
        hipEvent_t nccl_event;
        hipEventCreate(&nccl_event);
        hipEventCreateWithFlags(&gemm_event, hipEventBlockingSync);
        hipEventCreateWithFlags(&nccl_event, hipEventBlockingSync);

        for (size_t chunkStart = 0; chunkStart < M_N; chunkStart += chunkSize) {
            cublasProcessMatMulChunk(handle,
                                    cublasStream,
                                    chunkStart,
                                    chunkStart + chunkSize,
                                    &hAlpha,
                                    &hBeta,
                                    input,
                                    weight,
                                    output,
                                    M,
                                    N,
                                    K,
                                    column_parallel);
            hipEventRecord(gemm_event, cublasStream);
            hipStreamWaitEvent(ncclStream, gemm_event, 0);
            hipEventRecord(nccl_event, ncclStream);
            NCCLCHECK(ncclAllReduce(output + chunkStart,
                                    output + chunkStart,
                                    chunkSize,
                                    ncclFloat16,
                                    ncclSum,
                                    comm,
                                    ncclStream));
            hipStreamWaitEvent(cublasStream, nccl_event, 0);
        }

        hipEventRecord(nccl_event, ncclStream);
        if (!column_parallel) {
            hipStreamWaitEvent(ncclStream, nccl_event, 0);
            hipStreamWaitEvent(cublasStream, nccl_event, 0);
        }
        hipEventDestroy(nccl_event);
        hipEventDestroy(gemm_event);
        return 0;
    } else {
        printf("Failed: kernel error %s:%d '%s'\n", __FILE__, __LINE__, "The setting value " \
                "(tp_atten_parallel > 0 or tp_mlp_parallel > 0) " \
                "needs to be divisible by batch*seq_len*hidden");            
        exit(EXIT_FAILURE); 
    }
}

int pipe_overlapped_with_split_kernel(hipblasHandle_t handle,
                                      const float* input,
                                      const float* weight,
                                      float* output,
                                      int slit_n,
                                      int M,
                                      int N,
                                      int K,
                                      hipStream_t cublasStream,
                                      ncclComm_t comm,
                                      hipStream_t ncclStream,
                                      float alpha,
                                      float beta,
                                      bool column_parallel) {
    size_t M_N = static_cast<size_t>(M) * static_cast<size_t>(N);

    if (slit_n >= 0 && M_N % slit_n == 0) {
        
        size_t chunkSize = M_N / slit_n;
        hipEvent_t gemm_event;
        hipEventCreate(&gemm_event);
        hipEvent_t nccl_event;
        hipEventCreate(&nccl_event);
        hipEventCreateWithFlags(&gemm_event, hipEventBlockingSync);
        hipEventCreateWithFlags(&nccl_event, hipEventBlockingSync);

        for (size_t chunkStart = 0; chunkStart < M_N; chunkStart += chunkSize) {
            cublasProcessMatMulChunk(handle,
                                    cublasStream,
                                    chunkStart,
                                    chunkStart + chunkSize,
                                    &alpha,
                                    &beta,
                                    input,
                                    weight,
                                    output,
                                    M,
                                    N,
                                    K,
                                    column_parallel);
            hipEventRecord(gemm_event, cublasStream);
            hipStreamWaitEvent(ncclStream, gemm_event, 0);
            hipEventRecord(nccl_event, ncclStream);
            NCCLCHECK(ncclAllReduce(output + chunkStart,
                                    output + chunkStart,
                                    chunkSize,
                                    ncclFloat32,
                                    ncclSum,
                                    comm,
                                    ncclStream));
            hipStreamWaitEvent(cublasStream, nccl_event, 0);
        }

        hipEventRecord(nccl_event, ncclStream);
        if (!column_parallel) {
            hipStreamWaitEvent(ncclStream, nccl_event, 0);
            hipStreamWaitEvent(cublasStream, nccl_event, 0);
        }
        hipEventDestroy(nccl_event);
        hipEventDestroy(gemm_event);
        return 0;
    } else {
        printf("Failed: kernel error %s:%d '%s'\n", __FILE__, __LINE__, "The setting value " \
                "(tp_atten_parallel > 0 or tp_mlp_parallel > 0) " \
                "needs to be divisible by batch*seq_len*hidden");            
        exit(EXIT_FAILURE); 
    }
}

int pipe_overlapped_with_split_kernel(hipblasHandle_t handle,
                                      const double* input,
                                      const double* weight,
                                      double* output,
                                      int slit_n,
                                      int M,
                                      int N,
                                      int K,
                                      hipStream_t cublasStream,
                                      ncclComm_t comm,
                                      hipStream_t ncclStream,
                                      float alpha,
                                      float beta,
                                      bool column_parallel) {
    // TODO(lixiao31): support double date type

    return 0;
}

template <typename T>
int matmul_reduce_parallel_forward_cuda(at::Tensor input,
                                        T* weight,
                                        int in_features,
                                        int batch_size,
                                        int out_features,
                                        T* output,
                                        void* lt_workspace,
                                        ncclComm_t comm,
                                        hipStream_t nccl_stream,
                                        int opt_num,
                                        float alpha,
                                        float beta,
                                        bool column_parallel) {
    int status = 1;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipStream_t cublas_stream;
    hipblasGetStream(handle, &cublas_stream);

    status = pipe_overlapped_with_split_kernel(handle,
                                               input.data_ptr<T>(),
                                               weight,
                                               output,
                                               opt_num,
                                               batch_size,
                                               out_features,
                                               in_features,
                                               cublas_stream,
                                               comm,
                                               nccl_stream,
                                               alpha,
                                               beta,
                                               column_parallel);
    return status;
}

template int matmul_reduce_parallel_forward_cuda<at::Half>(
    at::Tensor input,
    at::Half* weight,
    int in_features,
    int batch_size,
    int out_features,
    at::Half* output,
    void* lt_workspace,
    ncclComm_t comm,
    hipStream_t nccl_stream,
    int opt_num,
    float alpha,
    float beta,
    bool column_parallel);

template int matmul_reduce_parallel_forward_cuda<float>(
    at::Tensor input,
    float* weight,
    int in_features,
    int batch_size,
    int out_features,
    float* output,
    void* lt_workspace,
    ncclComm_t comm,
    hipStream_t nccl_stream,
    int opt_num,
    float alpha,
    float beta,
    bool column_parallel);

template int matmul_reduce_parallel_forward_cuda<double>(
    at::Tensor input,
    double* weight,
    int in_features,
    int batch_size,
    int out_features,
    double* output,
    void* lt_workspace,
    ncclComm_t comm,
    hipStream_t nccl_stream,
    int opt_num,
    float alpha,
    float beta,
    bool column_parallel);
